
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel

__global__ void helloCUDA()
{
    printf("Hello!\n");
}

int main()
{
    // 1 block and 1 thread
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
